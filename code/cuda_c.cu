
#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
#include <valarray>

__global__
void addVec(float* x, float* y, float* res, int n) {
   int tid = blockIdx.x * blockDim.x + threadIdx.x;
   int gridsize = blockDim.x * gridDim.x;

   for(int i = tid; i < n; i += gridsize) { res[i] = x[i] + y[i]; }
}

int main() {
   const int N = 100000, nBlocks = 256, nThreads=128;
   std::valarray<float> x(N), y(N), res(N);
   float* x_d, * y_d, * res_d;
   int nBytes = N * sizeof(float);

   srand(1234);

   for(int i = 0; i < N; i++) {
      x[i] = float(rand()) / RAND_MAX;
      y[i] = float(rand()) / RAND_MAX;
   }

   hipMalloc(&x_d, N * sizeof(float));
   hipMalloc(&y_d, N * sizeof(float));
   hipMalloc(&res_d, N * sizeof(float));

   hipMemcpy(x_d, &x[0], nBytes, hipMemcpyHostToDevice);
   hipMemcpy(y_d, &y[0], nBytes, hipMemcpyHostToDevice);

   addVec<<<nBlocks, nThreads>>>(x_d, y_d, res_d, N);

   hipMemcpy(&res[0], res_d, nBytes, hipMemcpyDeviceToHost);

   std::cout << "Max error: "
             << std::abs(res - (x + y)).max()
             << std::endl;
   hipFree(x_d);
   hipFree(y_d);
   hipFree(res_d);
   return 0;
}
